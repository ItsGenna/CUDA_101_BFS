
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <sys/time.h>
#include <time.h>
#include <vector>

#define MAX_FRONTIER_SIZE 300 //before it was 128

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }


__global__ void BFS_CUDA(const int* rowPointers_d, const int* destinations_d,
                         int* distances_d, int* currentFrontier_d, int* currentFrontierSize_d,
                         int dim, int max_frontier_size) {
    extern __shared__ int shared_frontier[];  // Memoria condivisa per la frontiera locale
    __shared__ int shared_frontier_size;

    // Inizializza la dimensione della frontiera locale
    if (threadIdx.x == 0) {
        shared_frontier_size = 0;
    }

    __syncthreads();

    // Calcola l'indice globale del thread
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < dim) {
        // Ottieni il vertice corrente dalla frontiera
        int currentVertex = currentFrontier_d[i];
        printf("Thread %d processing vertex %d\n", i, currentVertex);

        // Itera sui vicini del vertice corrente
        int start = rowPointers_d[currentVertex];
        int end = rowPointers_d[currentVertex + 1];

        for (int j = start; j < end; ++j) {
            int neighbor = destinations_d[j];
            printf("Thread %d processing edge %d -> %d, j = %d\n", i, currentVertex, neighbor, j);

            // Aggiorna la distanza se il vicino non è ancora stato visitato
            if (atomicCAS(&distances_d[neighbor], -1, distances_d[currentVertex] + 1) == -1) {
                printf("Thread %d updating distance for vertex %d: %d\n", i, neighbor, distances_d[currentVertex] + 1);

                // Aggiungi il vicino alla frontiera locale
                int idx = atomicAdd(&shared_frontier_size, 1);
                if (idx < max_frontier_size) {
                    shared_frontier[idx] = neighbor;
                    printf("Thread %d added vertex %d to shared frontier at index %d\n", i, neighbor, idx);
                } else {
                    printf("Thread %d could not add vertex %d to shared frontier (overflow)\n", i, neighbor);
                }
            }
        }
    }
    __syncthreads();

    // Scrivi la frontiera locale nella memoria globale
    if (threadIdx.x == 0) {
        int global_idx = atomicExch(currentFrontierSize_d, shared_frontier_size);

        // Copia i nuovi vertici nella frontiera globale, evitando overflow
        for (int j = 0; j < shared_frontier_size; ++j) {
            if (global_idx + j < max_frontier_size) {
                currentFrontier_d[j] = shared_frontier[j];
                printf("Block %d added vertex %d to global frontier at index %d\n", blockIdx.x, shared_frontier[j], j);
            } else {
                printf("Block %d could not add vertex %d to global frontier (overflow)\n", blockIdx.x, shared_frontier[j]);
            }
        }
    }
}

void read_matrix(std::vector<int> &row_ptr,
                 std::vector<int> &col_ind,
                 std::vector<float> &values,
                 const std::string &filename,
                 int &num_rows,
                 int &num_cols,
                 int &num_vals);

void insertIntoFrontier(int val, int *frontier, int *frontier_size) {
    frontier[*frontier_size] = val;
    *frontier_size = *frontier_size + 1;
}

inline void swap(int **ptr1, int **ptr2) {
    int *tmp = *ptr1;
    *ptr1    = *ptr2;
    *ptr2    = tmp;
}


void BFS_parallel(const int source, const int* rowPointers, const int* destinations,
                  int* distances, const int num_rows, int num_vals) {
    int *currentFrontier_d, *currentFrontierSize_d;
    int *distances_d, *rowPointers_d, *destinations_d;

    // Allocazione memoria device
    CHECK(hipMalloc(&currentFrontier_d, MAX_FRONTIER_SIZE * sizeof(int)));
    CHECK(hipMalloc(&currentFrontierSize_d, sizeof(int)));
    CHECK(hipMalloc(&distances_d, num_rows * sizeof(int)));
    CHECK(hipMalloc(&rowPointers_d, num_rows * sizeof(int)));
    CHECK(hipMalloc(&destinations_d, num_vals * sizeof(int)));

    // Inizializzazione distanze e frontiera
    int initialDistances[num_rows];
    for (int i = 0; i < num_rows; ++i) {
        initialDistances[i] = -1;  // Inizializza tutte le distanze a -1
    }
    initialDistances[source] = 0;  // Nodo sorgente a distanza 0
    CHECK(hipMemcpy(distances_d, initialDistances, num_rows * sizeof(int), hipMemcpyHostToDevice));

    // Inizializza la frontiera
    int initialFrontierSize = 1;
    CHECK(hipMemcpy(currentFrontier_d, &source, sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(currentFrontierSize_d, &initialFrontierSize, sizeof(int), hipMemcpyHostToDevice));

    // Copia dati su device
    CHECK(hipMemcpy(rowPointers_d, rowPointers, num_rows * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(destinations_d, destinations, num_vals * sizeof(int), hipMemcpyHostToDevice));

    // Iterazione BFS
    while (initialFrontierSize > 0) {
        int blockDim = 256;
        int gridDim = (initialFrontierSize + blockDim - 1) / blockDim;
        int sharedMemSize = blockDim * sizeof(int);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error before kernel launch: %s\n", hipGetErrorString(err));
        }

        BFS_CUDA<<<gridDim, blockDim, sharedMemSize>>>(rowPointers_d, destinations_d, distances_d,
                                                       currentFrontier_d, currentFrontierSize_d,
                                                       initialFrontierSize, MAX_FRONTIER_SIZE);
        CHECK_KERNELCALL();

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error after kernel launch: %s\n", hipGetErrorString(err));
        }

        // Aggiorna la dimensione della frontiera
        CHECK(hipMemcpy(&initialFrontierSize, currentFrontierSize_d, sizeof(int), hipMemcpyDeviceToHost));

        // Debug: verificare la frontiera aggiornata
        int host_frontier[MAX_FRONTIER_SIZE];
        CHECK(hipMemcpy(host_frontier, currentFrontier_d, MAX_FRONTIER_SIZE * sizeof(int), hipMemcpyDeviceToHost));
        printf("Updated frontier: ");
        for (int i = 0; i < initialFrontierSize; ++i) {
            printf("%d ", host_frontier[i]);
        }
        printf("\n");

        // Reset dimensione frontiera per il prossimo passo
        CHECK(hipMemset(currentFrontierSize_d, 0, sizeof(int)));
    }

    // Copia risultato su host
    CHECK(hipMemcpy(distances, distances_d, num_rows * sizeof(int), hipMemcpyDeviceToHost));

    // Libera memoria
    hipFree(currentFrontier_d);
    hipFree(currentFrontierSize_d);
    hipFree(distances_d);
    hipFree(rowPointers_d);
    hipFree(destinations_d);
}


int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: ./exec matrix_file source\n");
        return 0;
    } // checks if the number of arguments passed is correct, otherwise returns an error message

    std::vector<int> row_ptr;
    std::vector<int> col_ind;
    std::vector<float> values;
    int num_rows, num_cols, num_vals;

    const std::string filename{argv[1]}; // argv[1] is the name of the file passed
    // The node starts from 1 but array starts from 0
    const int source = atoi(argv[2]) - 1; // source is the starting point of the algorithm (zero-indexed)

    read_matrix(row_ptr, col_ind, values, filename, num_rows, num_cols, num_vals);

    // Initialize dist to -1
    std::vector<int> dist(num_rows); //before it was num_vals
    for (int i = 0; i < num_rows; i++) { dist[i] = -1; }
    // Compute in sw
    BFS_parallel(source, row_ptr.data(), col_ind.data(), dist.data(), num_rows, num_vals); // .data() returns a pointer to the first element
    // of the array
    printf("\nFinal distances:\n");
    for (int i=0; i<num_rows; i++) {
        printf("%d ", dist[i]);
    }

    return EXIT_SUCCESS;
}

// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(std::vector<int> &row_ptr, // row_ptr will get filled with the row indexes of the array value
        // corresponding to the beginning of the new row
                 std::vector<int> &col_ind, // col_ind will get filled with the column indexes of the values
                 std::vector<float> &values, // values will get filled with the non-zero values of the matrix
                 const std::string &filename,
                 int &num_rows,
                 int &num_cols,
                 int &num_vals) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "File cannot be opened!\n";
        throw std::runtime_error("File cannot be opened");
    }

    // Get number of rows, columns, and non-zero values
    file >> num_rows >> num_cols >> num_vals;// these values are in the first row of the file

    row_ptr.resize(num_rows + 1); // changing the size of the arrays
    col_ind.resize(num_vals);
    values.resize(num_vals);

    // Collect occurrences of each row for determining the indices of row_ptr
    std::vector<int> row_occurrences(num_rows, 0);

    int row, column;
    float value;
    while (file >> row >> column >> value) {
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;

        row_occurrences[row]++;
    }

    // Set row_ptr
    int index = 0;
    for (int i = 0; i < num_rows; i++) {
        row_ptr[i] = index;
        index += row_occurrences[i];
    }
    row_ptr[num_rows] = num_vals;

    // Reset the file stream to read again from the beginning
    file.clear();
    file.seekg(0, std::ios::beg);

    // Read the first line again to skip it
    file >> num_rows >> num_cols >> num_vals;

    std::fill(col_ind.begin(), col_ind.end(), -1);

    int i = 0;
    while (file >> row >> column >> value) {
        row--;
        column--;

        // Find the correct index (i + row_ptr[row]) using both row information and an index i
        while (col_ind[i + row_ptr[row]] != -1) { i++; }
        col_ind[i + row_ptr[row]] = column;
        values[i + row_ptr[row]]  = value;
        i                         = 0;
    }
}