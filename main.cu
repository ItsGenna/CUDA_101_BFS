#include <cstdlib>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <sys/time.h>
#include <time.h>
#include <vector>
#include <hip/hip_runtime.h>


#define MAX_MATRIX_LENGTH 7500
#define MAX_FRONTIER_SIZE 256

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

__constant__ int destinations_constant[MAX_MATRIX_LENGTH];
__constant__ int rowPointers_constant[MAX_MATRIX_LENGTH];
__device__ int globalCounter;


__global__ void BFS_CUDA(int* distances_d, int* currentFrontier_d, int* currentFrontierSize_d, int dim, int currentDistance) {
    // reset global counter
    if (threadIdx.x == 0) {
        globalCounter = 0;
    }

    __syncthreads();

    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < dim) {
        int currentVertex = destinations_constant[i];
        printf("\nCurrent distance: %d, Current vertex: %d", currentDistance, currentVertex);
        int start = rowPointers_constant[currentVertex];
        int end = rowPointers_constant[currentVertex + 1];

        // check if it has already been visited and update the distance
        if (atomicCAS(&distances_d[currentVertex], -1, currentDistance) == -1) {
            int count = (int)(end - start) + 1;
            // if it hasn't been visited, we have to add it to the next iteration's frontier
            int offset = atomicAdd(&globalCounter, count); // first we update the counter that keeps track of the size of the frontier
            for (int j = 0; j < count; ++j) {
                currentFrontier_d[offset + j] = start + j; // then we change the frontier itself
            }
        }
    }

    if (threadIdx.x==0){
        *currentFrontierSize_d = globalCounter;
    }
}


void read_matrix(std::vector<int> &row_ptr,
                 std::vector<int> &col_ind,
                 std::vector<float> &values,
                 const std::string &filename,
                 int &num_rows,
                 int &num_cols,
                 int &num_vals);

void insertIntoFrontier(int val, int *frontier, int *frontier_size) {
    frontier[*frontier_size] = val;
    *frontier_size = *frontier_size + 1;
}

inline void swap(int **ptr1, int **ptr2) {
    int *tmp = *ptr1;
    *ptr1    = *ptr2;
    *ptr2    = tmp;
}


void BFS_parallel(const int source, const int* rowPointers, const int* destinations,
                  int* distances, const int num_rows, int num_vals) {
    int *currentFrontier_d, *currentFrontierSize_d;
    int *distances_d;

    // device memory allocation
    CHECK(hipMalloc(&currentFrontier_d, MAX_FRONTIER_SIZE * sizeof(int)));
    CHECK(hipMalloc(&currentFrontierSize_d, sizeof(int)));
    CHECK(hipMalloc(&distances_d, num_rows * sizeof(int)));

    // initialize
    int initialDistances[num_rows];
    for (int i = 0; i < num_rows; ++i) {
        initialDistances[i] = -1;  // all the distances have to be -1
    }
    initialDistances[source] = 0;  // except the starting value, which has to be 1

    // copy to device memory
    CHECK(hipMemcpy(distances_d, initialDistances, num_rows * sizeof(int), hipMemcpyHostToDevice));
    int initialFrontierSize = 1;
    CHECK(hipMemcpy(currentFrontier_d, &source, sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(currentFrontierSize_d, &initialFrontierSize, sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(rowPointers_constant), rowPointers, num_rows * sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(destinations_constant), destinations, num_vals * sizeof(int))); // copy to constant memory
    int currentDist = 0;

    while (initialFrontierSize > 0) {
        int block_Dim = 256;
        int grid_Dim = (initialFrontierSize + block_Dim - 1) / block_Dim;

        // launch the kernel
        BFS_CUDA<<<grid_Dim, block_Dim>>>(distances_d, currentFrontier_d,
                                                         currentFrontierSize_d,initialFrontierSize, currentDist);
        CHECK_KERNELCALL();

        // updates frontier dimensions
        CHECK(hipMemcpy(&initialFrontierSize, currentFrontierSize_d, sizeof(int), hipMemcpyDeviceToHost));


        // reset current frontier for next iteration, probably can be removed (check later)
        CHECK(hipMemset(currentFrontierSize_d, 0, sizeof(int)));
        currentDist ++;
    }

    // copy results to host
    CHECK(hipMemcpy(distances, distances_d, num_rows * sizeof(int), hipMemcpyDeviceToHost));

    // free device memory
    hipFree(currentFrontier_d);
    hipFree(currentFrontierSize_d);
    hipFree(distances_d);
}


int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: ./exec matrix_file source\n");
        return 0;
    } // checks if the number of arguments passed is correct, otherwise returns an error message

    std::vector<int> row_ptr;
    std::vector<int> col_ind;
    std::vector<float> values;
    int num_rows, num_cols, num_vals;

    const std::string filename{argv[1]}; // argv[1] is the name of the file passed
    // The node starts from 1 but array starts from 0
    const int source = atoi(argv[2]) - 1; // source is the starting point of the algorithm (zero-indexed)

    read_matrix(row_ptr, col_ind, values, filename, num_rows, num_cols, num_vals);

    if (num_vals>MAX_MATRIX_LENGTH){
        printf("Matrix is too big");
        return 0;
    }

    // Initialize dist to -1
    std::vector<int> dist(num_rows); //before it was num_vals
    for (int i = 0; i < num_rows; i++) { dist[i] = -1; }

    clock_t start, end;
    start = clock();

    BFS_parallel(source, row_ptr.data(), col_ind.data(), dist.data(), num_rows, num_vals); // .data() returns a pointer to the first element
    // of the array

    end = clock();

    printf("\nFinal distances:\n");
    for (int i=0; i<num_rows; i++) {
        printf("%d ", dist[i]);
    }

    printf("\nTime elapsed: %f ms", float(end-start)*1000/CLOCKS_PER_SEC);

    return EXIT_SUCCESS;
}

// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(std::vector<int> &row_ptr, // row_ptr will get filled with the row indexes of the array value
        // corresponding to the beginning of the new row
                 std::vector<int> &col_ind, // col_ind will get filled with the column indexes of the values
                 std::vector<float> &values, // values will get filled with the non-zero values of the matrix
                 const std::string &filename,
                 int &num_rows,
                 int &num_cols,
                 int &num_vals) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "File cannot be opened!\n";
        throw std::runtime_error("File cannot be opened");
    }

    // Get number of rows, columns, and non-zero values
    file >> num_rows >> num_cols >> num_vals;// these values are in the first row of the file

    row_ptr.resize(num_rows + 1); // changing the size of the arrays
    col_ind.resize(num_vals);
    values.resize(num_vals);

    // Collect occurrences of each row for determining the indices of row_ptr
    std::vector<int> row_occurrences(num_rows, 0);

    int row, column;
    float value;
    while (file >> row >> column >> value) {
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;

        row_occurrences[row]++;
    }

    // Set row_ptr
    int index = 0;
    for (int i = 0; i < num_rows; i++) {
        row_ptr[i] = index;
        index += row_occurrences[i];
    }
    row_ptr[num_rows] = num_vals;

    // Reset the file stream to read again from the beginning
    file.clear();
    file.seekg(0, std::ios::beg);

    // Read the first line again to skip it
    file >> num_rows >> num_cols >> num_vals;

    std::fill(col_ind.begin(), col_ind.end(), -1);

    int i = 0;
    while (file >> row >> column >> value) {
        row--;
        column--;

        // Find the correct index (i + row_ptr[row]) using both row information and an index i
        while (col_ind[i + row_ptr[row]] != -1) { i++; }
        col_ind[i + row_ptr[row]] = column;
        values[i + row_ptr[row]]  = value;
        i                         = 0;
    }
}