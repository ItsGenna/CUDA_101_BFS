#include <cstdlib>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <sys/time.h>
#include <time.h>
#include <vector>
#include <hip/hip_runtime.h>


#define MAX_MATRIX_LENGTH 7500
#define MAX_FRONTIER_SIZE 5000

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

__constant__ int destinations_constant[MAX_MATRIX_LENGTH];
__constant__ int rowPointers_constant[MAX_MATRIX_LENGTH];


__global__ void BFS_CUDA(int* distances_d, const int* currentFrontier_d, int* nextFrontier, int* frontierSize_d, int dim, int currentDistance) {
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < dim) {
        int currentVertex = currentFrontier_d[i];
        int start = rowPointers_constant[currentVertex];
        int end = rowPointers_constant[currentVertex + 1];

        // check if it has already been visited and update the distance
        if (atomicCAS(&distances_d[currentVertex], -1, currentDistance) == -1) {
            int count = (int)(end - start);
            // if it hasn't been visited, we have to add it to the next iteration's frontier
            int offset = atomicAdd(frontierSize_d, count); // first we update the counter that keeps track of the size of the frontier
            for (int j = 0; j < count; ++j) {
                nextFrontier[offset + j] = destinations_constant[start+j]; // then we change the frontier itself
            }
        }
    }
}


void read_matrix(std::vector<int> &row_ptr,
                 std::vector<int> &col_ind,
                 std::vector<float> &values,
                 const std::string &filename,
                 int &num_rows,
                 int &num_cols,
                 int &num_vals);

void insertIntoFrontier(int val, int *frontier, int *frontier_size) {
    frontier[*frontier_size] = val;
    *frontier_size = *frontier_size + 1;
}

inline void swap(int **ptr1, int **ptr2) {
    int *tmp = *ptr1;
    *ptr1    = *ptr2;
    *ptr2    = tmp;
}


void BFS_parallel(const int source, const int* rowPointers, const int* destinations,
                  int* distances, const int num_rows, int num_vals) {
    int *currentFrontier_d, *currentFrontierSize_d, *nextFrontier_d;
    int *distances_d;

    // device memory allocation
    CHECK(hipMalloc(&currentFrontier_d, MAX_FRONTIER_SIZE * sizeof(int)));
    CHECK(hipMalloc(&nextFrontier_d, MAX_FRONTIER_SIZE * sizeof(int)));
    CHECK(hipMalloc(&currentFrontierSize_d, sizeof(int)));
    CHECK(hipMalloc(&distances_d, num_rows * sizeof(int)));

    // initialize
    int initialDistances[num_rows];
    for (int i = 0; i < num_rows; ++i) {
        initialDistances[i] = -1;  // all the distances have to be -1
    }
    initialDistances[source] = 0;  // except the starting value, which has to be 1

    // handle first iteration
    int start_host = rowPointers[source];
    int end_host = rowPointers[source + 1];
    int initialFrontierSize = end_host - start_host;
    int initialFrontier[initialFrontierSize];

    for (int j = 0; j < initialFrontierSize; ++j) {
        int neighbor = destinations[start_host + j];
           initialFrontier[j] = neighbor;
    }

    // copy to device memory
    CHECK(hipMemcpy(distances_d, initialDistances, num_rows * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(currentFrontier_d, initialFrontier, initialFrontierSize*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(currentFrontierSize_d, &initialFrontierSize, sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(rowPointers_constant), rowPointers, (num_rows+1) * sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(destinations_constant), destinations, num_vals * sizeof(int))); // copy to constant memory
    int currentDist = 1;

    while (initialFrontierSize > 0) {
        int block_Dim = 256;
        int grid_Dim = (initialFrontierSize + block_Dim - 1) / block_Dim;

        // launch the kernel
        BFS_CUDA<<<grid_Dim, block_Dim>>>(distances_d, currentFrontier_d, nextFrontier_d,
                                          currentFrontierSize_d,initialFrontierSize, currentDist);
        CHECK_KERNELCALL();

        // updates frontier dimensions
        CHECK(hipMemcpy(&initialFrontierSize, currentFrontierSize_d, sizeof(int), hipMemcpyDeviceToHost));


        // reset current frontier for next iteration, probably can be removed (check later)
        CHECK(hipMemset(currentFrontierSize_d, 0, sizeof(int)));
        swap(&currentFrontier_d, &nextFrontier_d);

        currentDist ++;
    }

    // copy results to host
    CHECK(hipMemcpy(distances, distances_d, num_rows * sizeof(int), hipMemcpyDeviceToHost));

    // free device memory
    hipFree(currentFrontier_d);
    hipFree(currentFrontierSize_d);
    hipFree(distances_d);
    hipFree(nextFrontier_d);
}


int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: ./exec matrix_file source\n");
        return 0;
    } // checks if the number of arguments passed is correct, otherwise returns an error message

    std::vector<int> row_ptr;
    std::vector<int> col_ind;
    std::vector<float> values;
    int num_rows, num_cols, num_vals;

    const std::string filename{argv[1]}; // argv[1] is the name of the file passed
    // The node starts from 1 but array starts from 0
    const int source = atoi(argv[2]) - 1; // source is the starting point of the algorithm (zero-indexed)

    read_matrix(row_ptr, col_ind, values, filename, num_rows, num_cols, num_vals);

    if (num_vals>MAX_MATRIX_LENGTH){
        printf("Matrix is too big");
        return 0;
    }

    // Initialize dist to -1
    std::vector<int> dist(num_rows); //before it was num_vals
    for (int i = 0; i < num_rows; i++) { dist[i] = -1; }

    clock_t start, end;
    start = clock();

    BFS_parallel(source, row_ptr.data(), col_ind.data(), dist.data(), num_rows, num_vals); // .data() returns a pointer to the first element
    // of the array

    end = clock();

    printf("\nFinal distances:\n");
    for (int i=0; i<num_rows; i++) {
        printf("%d ", dist[i]);
    }

    printf("\nTime elapsed: %f ms", float(end-start)*1000/CLOCKS_PER_SEC);

    return EXIT_SUCCESS;
}

// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(std::vector<int> &row_ptr, // row_ptr will get filled with the row indexes of the array value
        // corresponding to the beginning of the new row
                 std::vector<int> &col_ind, // col_ind will get filled with the column indexes of the values
                 std::vector<float> &values, // values will get filled with the non-zero values of the matrix
                 const std::string &filename,
                 int &num_rows,
                 int &num_cols,
                 int &num_vals) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "File cannot be opened!\n";
        throw std::runtime_error("File cannot be opened");
    }

    // Get number of rows, columns, and non-zero values
    file >> num_rows >> num_cols >> num_vals;// these values are in the first row of the file

    row_ptr.resize(num_rows + 1); // changing the size of the arrays
    col_ind.resize(num_vals);
    values.resize(num_vals);

    // Collect occurrences of each row for determining the indices of row_ptr
    std::vector<int> row_occurrences(num_rows, 0);

    int row, column;
    float value;
    while (file >> row >> column >> value) {
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;

        row_occurrences[row]++;
    }

    // Set row_ptr
    int index = 0;
    for (int i = 0; i < num_rows; i++) {
        row_ptr[i] = index;
        index += row_occurrences[i];
    }
    row_ptr[num_rows] = num_vals;

    // Reset the file stream to read again from the beginning
    file.clear();
    file.seekg(0, std::ios::beg);

    // Read the first line again to skip it
    file >> num_rows >> num_cols >> num_vals;

    std::fill(col_ind.begin(), col_ind.end(), -1);

    int i = 0;
    while (file >> row >> column >> value) {
        row--;
        column--;

        // Find the correct index (i + row_ptr[row]) using both row information and an index i
        while (col_ind[i + row_ptr[row]] != -1) { i++; }
        col_ind[i + row_ptr[row]] = column;
        values[i + row_ptr[row]]  = value;
        i                         = 0;
    }
}